#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#include "kernel.cu"
#include "dev_array.h"
#include <math.h>

using namespace std;

class Timer {
    struct timespec s_;
public:
    Timer() { tic(); }
    void tic() {
        clock_gettime(CLOCK_REALTIME, &s_);
    }

    double toc() {
        struct timespec e;
        clock_gettime(CLOCK_REALTIME, &e);
        return (double)(e.tv_sec - s_.tv_sec) + 1e-9 * (double)(e.tv_nsec - s_.tv_nsec);
    }
};

int main()
{
    int N = 2048;
    int SIZE = N*N;
    Timer t;
    double elapsed;
    int iterations = 1;
    // Allocate memory on the host
    vector<float> h_A(SIZE);
    vector<float> h_B(SIZE);
    vector<float> h_C(SIZE);

    // Initialize matrices on the host
    srand(time(0));
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            h_A[i*N+j] = rand() % 100;
            h_B[i*N+j] = rand() % 100;
        }
    }

    // Allocate memory on the device
    dev_array<float> d_A(SIZE);
    dev_array<float> d_B(SIZE);
    dev_array<float> d_C(SIZE);

    d_A.set(&h_A[0], SIZE);
    d_B.set(&h_B[0], SIZE);

    for (int i = 0; i < iterations; i++) {
      t.tic();
      matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
      elapsed += t.toc();
    }
    printf("cuda mat mul %lf ms\n", 1000.0 * elapsed / iterations);
    hipDeviceSynchronize();

    d_C.get(&h_C[0], SIZE);
    hipDeviceSynchronize();

    return 0;
}